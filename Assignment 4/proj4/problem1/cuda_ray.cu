#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

#define SPHERES 20
#define BLOCK_SIZE 32
#define GRID_SIZE 64

#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

using namespace std;
using namespace std::chrono;

struct Sphere {
    float r, b, g;
    float radius;
    float x, y, z;

    // hit function will be executed in gpu
    __device__ float hit(float ox, float oy, float *n) {
        float dx = ox - this->x;
        float dy = oy - this->y;
        if (dx * dx + dy * dy < this->radius * this->radius) {
            float dz = sqrtf(this->radius * this->radius - dx * dx - dy * dy);
            *n = dz / sqrtf(this->radius * this->radius);
            return dz + this->z;
        }
        return -INF;
    }
};

extern const uint3 threadIdx, blockIdx;

__global__ void kernel(Sphere *s, unsigned char *ptr) {
    int tx = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    int ty = threadIdx.y + blockIdx.y * BLOCK_SIZE;
    int tid = tx + ty * DIM;
    float ox = (tx - DIM / 2);
    float oy = (ty - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[tid * 4 + 0] = (int)(r * 255);
    ptr[tid * 4 + 1] = (int)(g * 255);
    ptr[tid * 4 + 2] = (int)(b * 255);
    ptr[tid * 4 + 3] = 255;
}

void ppm_write(unsigned char *bitmap, int xdim, int ydim, FILE *fp) {
    int i, x, y;
    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");
    for (y = 0; y < ydim; y++) {
        for (x = 0; x < xdim; x++) {
            i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}

int main(void) {
    FILE *fp;
    Sphere *temp_s, *dev_temp_s;
    unsigned char *bitmap, *dev_bitmap;
    // dimension size of cuda processing
    dim3 grid_size = { GRID_SIZE, GRID_SIZE, 1 };
    dim3 block_size = { BLOCK_SIZE, BLOCK_SIZE, 1 };

    srand(time(NULL));

    fp = fopen("result.ppm", "w");
    temp_s = (Sphere *)malloc(sizeof(Sphere) * SPHERES);
    for (int i = 0; i < SPHERES; i++) {
        temp_s[i].r = rnd(1.0f), temp_s[i].g = rnd(1.0f), temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(2000.0f) - 1000, temp_s[i].y = rnd(2000.0f) - 1000, temp_s[i].z = rnd(2000.0f) - 1000;
        temp_s[i].radius = rnd(200.0f) + 40;
    }
    bitmap = (unsigned char *)malloc(sizeof(unsigned char) * DIM * DIM * 4);

    auto start_t = high_resolution_clock::now();
    hipMalloc((void**)&dev_temp_s, sizeof(Sphere) * SPHERES);
    hipMalloc((void**)&dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4);
    hipMemcpy(dev_temp_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);
    kernel<<<grid_size, block_size>>>(dev_temp_s, dev_bitmap);
    hipMemcpy(bitmap, dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);
    auto interval = high_resolution_clock::now() - start_t;
    
    ppm_write(bitmap, DIM, DIM, fp);
    fclose(fp);

    hipFree(dev_temp_s); hipFree(dev_bitmap);
    free(bitmap); free(temp_s);

    cout << "CUDA ray tracing: " << duration_cast<milliseconds>(interval).count() << " millisec\n";
    cout << "[result.ppm] was generated.\n";

    return 0;
}